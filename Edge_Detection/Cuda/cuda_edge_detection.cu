#include "hip/hip_runtime.h"
// Includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#define block_size 16

const char *imageFilename = "lena_bw_double.pgm";

// Texture memory declaration
texture<float, 2, hipReadModeElementType> imgTex;

//Constant memory variables declaration
__constant__ int constWidth[1];
__constant__ int constHeight[1];
__constant__ int constFilterSize[1];
__constant__ int constPadding[1];
__constant__ float constFilter[9];

//Application of filter to image
__global__ void convolution(float *oData){
  unsigned int r = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int c = blockIdx.y * blockDim.x + threadIdx.y;

  float sum = 0;

  if((r >= constPadding[0] && r < constHeight[0]-constPadding[0]) && (c >= constPadding[0] && c < constHeight[0]-constPadding[0]))
    sum = 0;
    for(int x = 0; x<constFilterSize[0]; x++)
    	for(int y = 0; y<constFilterSize[0]; y++)
        sum += constFilter[y + x*constFilterSize[0]]*tex2D(imgTex, (r+(y-constPadding[0])), c+(x-constPadding[0]));
    if(sum < 0)
      oData[c*constWidth[0]+r] = 0;
    else if (sum > 1)
      oData[c*constWidth[0]+r] = 1;
    else
      oData[c*constWidth[0]+r] = sum;
}

//This method pads and unpads the image
void padData(float* iData, float* oData, int paddedWidth, size_t n, const int padding, bool pad){
  int paddedEnds  = (paddedWidth * padding) + padding;

  int j = 0;
  for (size_t i = paddedEnds; i < n-paddedEnds; i++) {
    int mod = i%paddedWidth;
    if(mod >= padding && mod < paddedWidth-padding){
      if(pad)
        oData[i] = iData[j];
      else
        oData[j] = iData[i];
      j++;
    }
  }
}

int main(int argc, char **argv){
  //Starts overhead timer
  StopWatchInterface *overhead = NULL;
  sdkCreateTimer(&overhead);
  sdkStartTimer(&overhead);

  //Checks if file exists
  char *imagePath = sdkFindFilePath(imageFilename, "");
  if (imagePath == NULL){
      printf("Unable to source image file: %s\n", imageFilename);
      exit(EXIT_FAILURE);
  }

  unsigned int width, height;
  //initializes needed variables
  int filterSize[1] = {3};
  int padding[1] = {(filterSize[0]-1)/2};

  int filLength = pow(filterSize[0], 2);
  float hfilter[9] = {-1, 0, 1, -2, 0, 2, -1, 0, 1 };

  float *hData = 0;
  float *dData = 0;
  float *dOutputData = 0;
  float *filter = 0;
  char outputFilename[1024];

  sdkLoadPGM(imagePath, &hData, &width, &height);

  //Gets padded data set up
  int paddedWidth[1]      = {width + (2 * padding[0])};
  int paddedHeight[1]     = {height + (2 * padding[0])};
  size_t paddedElements   = paddedWidth[0] * paddedHeight[0];
  unsigned int paddedSize = paddedElements * sizeof(float);

  float *hOutputData = (float *)malloc(paddedSize);
  float *hpData      = (float *)malloc(paddedSize);
  hipMalloc((void **) &dData, paddedSize);
  hipMalloc((void **) &dOutputData, paddedSize);
  hipMalloc((void **) &filter, filLength*sizeof(float));

  padData(hData, hpData, paddedWidth[0], paddedElements, padding[0], true);

  hipMemcpy(dData, hpData, paddedSize, hipMemcpyHostToDevice);
  hipMemcpy(filter, hfilter, filLength*sizeof(float), hipMemcpyHostToDevice);

  //Copies variables to constant memory
  hipMemcpyToSymbol(HIP_SYMBOL(constWidth), paddedWidth, sizeof(int), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(constHeight), paddedHeight, sizeof(int), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(constFilterSize), filterSize, sizeof(int), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(constFilter), hfilter, filLength*sizeof(float), 0, hipMemcpyHostToDevice);

  // Allocate array and copy image data
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  hipArray *cuArray;
  hipMallocArray(&cuArray, &channelDesc, paddedWidth[0], paddedHeight[0]);
  hipMemcpyToArray(cuArray, 0, 0, hpData, paddedSize, hipMemcpyHostToDevice);
  imgTex.filterMode = hipFilterModePoint;
  hipBindTextureToArray(imgTex, cuArray, channelDesc);

  dim3 dimBlock(block_size, block_size, 1);
  dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);

  //Starts processing timer
  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);
  sdkStartTimer(&timer);

  convolution<<<dimGrid, dimBlock>>>(dOutputData);
  hipDeviceSynchronize();

  sdkStopTimer(&timer);
  printf("%f\n", sdkGetTimerValue(&timer));

  hipMemcpy(hOutputData, dOutputData, paddedSize, hipMemcpyDeviceToHost);

  padData(hOutputData, hData, paddedWidth[0], paddedElements, padding[0], false);

  strcpy(outputFilename, imagePath);
  strcpy(outputFilename + strlen(outputFilename) - 4, "_out.pgm");
  sdkSavePGM(outputFilename, hData, width, height);

  free(hData);
  free(hOutputData);
  hipFree(dData);
  hipFree(dOutputData);
  hipFree(filter);

  sdkStopTimer(&overhead);
  printf("%f\n", sdkGetTimerValue(&overhead)-sdkGetTimerValue(&timer));
  sdkDeleteTimer(&timer);
  sdkDeleteTimer(&overhead);

  return 0;
}
